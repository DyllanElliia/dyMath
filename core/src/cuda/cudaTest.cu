#include "hip/hip_runtime.h"
#include"./cudaTest.hpp"

__global__ void Decrease2(int *a, int *b, int *c) { *c = *a - *b; }
void addWithCuda2(int *c, int *a, int *b) {
  int *dev_c = 0;
  int *dev_a = 0;
  int *dev_b = 0;
  // 3.请求CUDA设备的内存（显存），执行CUDA函数
  hipMalloc((void **)&dev_c, sizeof(int));
  hipMalloc((void **)&dev_a, sizeof(int));
  hipMalloc((void **)&dev_b, sizeof(int));

  // 4.从主机复制数据到设备上
  hipMemcpy(dev_a, a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeof(int), hipMemcpyHostToDevice);

  Decrease2<<<1, 1>>>(dev_a, dev_b, dev_c);

  // 5.等待设备所有线程任务执行完毕
  hipDeviceSynchronize();

  // 6.数据复制到主机，释放占用空间
  hipMemcpy(c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipDeviceReset();         // 7.重置CUDA设备释放程序占用的资源
}